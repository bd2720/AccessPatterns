#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include "access.h"

// number of blocks (8 optimal)
#define GRID_DIM 8
// number of threads per block (64 optimal)
#define BLOCK_DIM 64
// total number of gpu threads
#define NTHREADS (GRID_DIM * BLOCK_DIM)
#define CUDA_CHECK() assert(hipGetLastError() == hipSuccess)

int arrChunked[ARR_SIZE];
int arrStriped[ARR_SIZE];

/*
	Unbalanced, old version of chunked:
	The final thread may execute half the work, in the worst case.
*/
__global__ void chunkedLazy(int *arr_d){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int i = (ARR_SIZE / NTHREADS) * id;
	int iMax;
	if(id == NTHREADS - 1){
		iMax = ARR_SIZE;
	} else {
		iMax = i + (ARR_SIZE / NTHREADS);
	}
	while(i < iMax){
		arr_d[i] = id;
		i++;
	}
}
/*
	Balanced version of chunked:
	(ARR_SIZE % NTHREADS) threads make ((ARR_SIZE / NTHREADS) + 1) access,
	while the rest make (ARR_SIZE / NTHREADS) accesses.
*/
__global__ void chunked(int *arr_d){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int i; // start
	int iMax; // end (+ 1)
	if(id < (ARR_SIZE % NTHREADS)){ // do 1 extra
		i = ((ARR_SIZE / NTHREADS) + 1) * id;
		iMax = i + ((ARR_SIZE / NTHREADS) + 1);
	} else { // don't do extra
		i = ((ARR_SIZE / NTHREADS) * id) + (ARR_SIZE % NTHREADS);
		iMax = i + (ARR_SIZE / NTHREADS);
	}
	while(i < iMax){
		arr_d[i] = id;
		i++;
	}
}

__global__ void striped(int *arr_d){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int i = id;
	while(i < ARR_SIZE){
		arr_d[i] = id;
		i += NTHREADS;
	}
}

int main(){
	printf("Running access-gpu...\n");
	printf("(GRID_DIM=%d) (BLOCK_DIM=%d) (ARR_SIZE=%d)\n", GRID_DIM, BLOCK_DIM, ARR_SIZE);
	printf("\n");

	struct timeval t_0, t_f;
	double tChunked, tStriped;
	
	int *arrChunked_d, *arrStriped_d;
	int size = ARR_SIZE * sizeof(int);
	hipMalloc((void **) &arrChunked_d, size);
	CUDA_CHECK();
	hipMalloc((void **) &arrStriped_d, size);
	CUDA_CHECK();
	
	// bad gpu access
	printf("Begin chunked access...\n");
	gettimeofday(&t_0, 0);
	chunked<<<GRID_DIM, BLOCK_DIM>>>(arrChunked_d);
	CUDA_CHECK();
	hipDeviceSynchronize();
	CUDA_CHECK();
	gettimeofday(&t_f, 0);
	tChunked = TIME(t_0, t_f);
	printf("Chunked access complete in %lfs.\n", tChunked);
	printf("\n");

	// good gpu access
	printf("Begin striped access...\n");
	gettimeofday(&t_0, 0);
	striped<<<GRID_DIM, BLOCK_DIM>>>(arrStriped_d);
	CUDA_CHECK();
	hipDeviceSynchronize();
	CUDA_CHECK();
	gettimeofday(&t_f, 0);
	tStriped = TIME(t_0, t_f);
	printf("Striped access complete in %lfs.\n", tStriped);
	printf("\n");
	
	printf("tChunked/tStriped: %lf\n", tChunked / tStriped);
	printf("\n");


	hipMemcpy(arrChunked, arrChunked_d, size, hipMemcpyDeviceToHost);
	CUDA_CHECK();
	hipMemcpy(arrStriped, arrStriped_d, size, hipMemcpyDeviceToHost);
	CUDA_CHECK();
	
	hipFree(arrChunked_d);
	CUDA_CHECK();
	hipFree(arrStriped_d);
	CUDA_CHECK();

	// arrays are now available on host
	
	if(PRINTING){
		printArr(arrChunked);
		printArr(arrStriped);
	}

	return 0;
}
